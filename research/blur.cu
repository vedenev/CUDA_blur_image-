#include "hip/hip_runtime.h"
//tutorial: https://medium.com/@akshathvarugeese/cuda-c-functions-in-python-through-dll-and-ctypes-for-windows-os-c29f56361089
//how to make cross-platform code for shared library https://stackoverflow.com/questions/2164827/explicitly-exporting-shared-library-functions-in-linux


#if defined(_MSC_VER)
    //  Microsoft 
    #define EXPORT extern "C" __declspec(dllexport)
    #define IMPORT extern "C" __declspec(dllimport)
#elif defined(__GNUC__)
    //  GCC
    #define EXPORT __attribute__((visibility("default")))
    #define IMPORT
#else
    //  do nothing and hope for the best?
    #define EXPORT
    #define IMPORT
    #pragma warning Unknown dynamic link import/export semantics.
#endif

#include <iostream>

#include <hip/hip_runtime.h>

#include <cmath>

#include <hip/hip_runtime_api.h>

# define M_PI           3.14159265358979323846  /* pi */

const int N_CANNELS = 3;



const unsigned int core_gpu_max_size = 64 * 64;
__constant__ unsigned short core_gpu[core_gpu_max_size];

__global__ void blurKernel(unsigned char* image_gpu, int n_channels, int image_size_x, int image_strade, int kernel_size, unsigned char* image_out, int image_out_size_x, int image_out_size_y, int image_out_strade)
{
	
	const unsigned int index_output_y = blockIdx.y * blockDim.y + threadIdx.y;
	if (index_output_y < image_out_size_y)
	{
		const unsigned int index_output_x = blockIdx.x * blockDim.x + threadIdx.x;
		if (index_output_x < image_out_size_x)
		{
			const unsigned int index_output_channel = blockIdx.z * blockDim.z + threadIdx.z;
			int index_input_offset = image_strade * index_output_channel + index_output_y * image_size_x + index_output_x;
			unsigned short sum = 0;
			int core_index = 0;
			for (int y = 0; y < kernel_size; y++)
			{
				int index_input_offset_2 = index_input_offset + y * image_size_x;
				for (int x = 0; x < kernel_size; x++)
				{
					int index_input = index_input_offset_2 + x;
					sum += ((unsigned short)image_gpu[index_input]) * core_gpu[core_index];
					core_index++;
				}
			}
			int index_output = image_out_strade * index_output_channel + index_output_y * image_out_size_x + index_output_x;
			sum += 128; // for rounding
			image_out[index_output] = (unsigned char)(sum / 256);
		}
	}
}

int integer_division_ceiling(int x, int y)
{
	// https://stackoverflow.com/questions/2745074/fast-ceiling-of-an-integer-division-in-c-c
	return x / y + (x % y != 0);
}

EXPORT void blur(unsigned char* image, int image_size_x, int image_size_y, int kernel_size, float sigma, unsigned char* image_out) {

	// numpy indexing [channel_index, y_index, x_index]

	// create core:
	float mean = (kernel_size - 1.0) / 2.0;
	float variance = sigma * sigma;
	float variance2 = 2 * variance;
	float variance2pi = variance2 * M_PI;
	int kernel_size2 = kernel_size * kernel_size;
	float* core_float_cpu = (float*)malloc(sizeof(float) * kernel_size2);
	int index = 0;
	float core_float_cpu_sum = 0.0;
	for (int y = 0; y < kernel_size; y++)
	{
		float dy = (float)y - mean;
		float dy2 = dy * dy;
		for (int x = 0; x < kernel_size; x++)
		{
			float dx = (float)x - mean;
			float dx2 = dx * dx;

			float argument_tmp = (dx2 + dy2) / variance2;
			
			core_float_cpu[index] = exp(-argument_tmp) / (variance2pi);
			core_float_cpu_sum += core_float_cpu[index];
			index++;
		}
	}

	for (index = 0; index < kernel_size2; index++)
	{
		core_float_cpu[index] = core_float_cpu[index] / core_float_cpu_sum;
	}

	unsigned short* core_cpu;
	core_cpu = (unsigned short*)malloc(sizeof(unsigned short) * kernel_size2);


	for (index = 0; index < kernel_size2; index++)
	{
		core_cpu[index] = (unsigned short)(roundf(256.0 * core_float_cpu[index]));
	}


	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(core_gpu), core_cpu, sizeof(unsigned short) * kernel_size2));

	int image_size = image_size_x * image_size_y * N_CANNELS;
	unsigned char* image_gpu;
	if (hipMalloc((void**)&image_gpu, sizeof(unsigned char) * image_size) != hipSuccess)
	{
		std::cout << "Error allocating GPU image_gpu\n";
	}
	hipMemcpy(image_gpu, image, sizeof(unsigned char) * image_size, hipMemcpyHostToDevice);
	
	int image_out_size_x = image_size_x - kernel_size + 1;
	int image_out_size_y = image_size_y - kernel_size + 1;
	int image_out_size = image_out_size_x * image_out_size_y * N_CANNELS;
	unsigned char* image_out_gpu;
	if (hipMalloc((void**)&image_out_gpu, sizeof(unsigned char) * image_out_size) != hipSuccess)
	{
		std::cout << "Error allocating GPU image_out_gpu\n";
	}

	int image_out_thread_size_x = 16;
	int image_out_thread_size_y = 16;
	int image_out_thread_size_z = 1;
	int image_out_grid_size_x = integer_division_ceiling(image_out_size_x, image_out_thread_size_x);
	int image_out_grid_size_y = integer_division_ceiling(image_out_size_y, image_out_thread_size_y);
	dim3 grid_image_out(image_out_grid_size_x, image_out_grid_size_y, image_out_thread_size_z);
	dim3 threadBlock_image_out(image_out_thread_size_x, image_out_thread_size_y, N_CANNELS);
	int image_strade = image_size_x * image_size_y;
	int image_out_strade = image_out_size_x * image_out_size_y;
	blurKernel<<<grid_image_out, threadBlock_image_out>>>(image_gpu, N_CANNELS, image_size_x, image_strade,  kernel_size, image_out_gpu, image_out_size_x, image_out_size_y, image_out_strade);
	hipMemcpy(image_out, image_out_gpu, sizeof(char) * image_out_size, hipMemcpyDeviceToHost);
	
}